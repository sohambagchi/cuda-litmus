#include "hip/hip_runtime.h"
#include <iostream>
#include "litmus.cuh"
#include "functions.cu"

__global__ void litmus_test(
  d_atomic_uint* test_locations,
  ReadResults* read_results,
  uint* shuffled_workgroups,
  cuda::atomic<uint, cuda::thread_scope_device>* barrier,
  uint* scratchpad,
  uint* scratch_locations,
  KernelParams* kernel_params,
  TestInstance* test_instances) {

  uint shuffled_workgroup = shuffled_workgroups[blockIdx.x];
  if (shuffled_workgroup < kernel_params->testing_workgroups) {

#ifdef ACQUIRE
    cuda::memory_order thread_0_store = cuda::memory_order_release;
    cuda::memory_order thread_1_load = cuda::memory_order_acquire;
    cuda::memory_order thread_1_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_2_load = cuda::memory_order_acquire;
    #define FENCE_0()
    #define FENCE_1()
    #define FENCE_2()
#elif defined(RELEASE)
    cuda::memory_order thread_0_store = cuda::memory_order_release;
    cuda::memory_order thread_1_load = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_store = cuda::memory_order_release;
    cuda::memory_order thread_2_load = cuda::memory_order_acquire;
    #define FENCE_0()
    #define FENCE_1()
    #define FENCE_2()
#elif defined(RELAXED)
    cuda::memory_order thread_0_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_load = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_2_load = cuda::memory_order_relaxed;
    #define FENCE_0()
    #define FENCE_1()
    #define FENCE_2()
#elif defined(ALL_FENCE)
    cuda::memory_order thread_0_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_load = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_2_load = cuda::memory_order_relaxed;
    #define FENCE_0() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE_1() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE_2() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
#elif defined(THREAD_0_FENCE_ACQ)
    cuda::memory_order thread_0_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_load = cuda::memory_order_acquire;
    cuda::memory_order thread_1_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_2_load = cuda::memory_order_acquire;
    #define FENCE_0() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE_1()
    #define FENCE_2()
#elif defined(THREAD_0_FENCE_REL)
    cuda::memory_order thread_0_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_load = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_store = cuda::memory_order_release;
    cuda::memory_order thread_2_load = cuda::memory_order_acquire;
    #define FENCE_0() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE_1()
    #define FENCE_2()
#elif defined(THREAD_1_FENCE)
    cuda::memory_order thread_0_store = cuda::memory_order_release;
    cuda::memory_order thread_1_load = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_2_load = cuda::memory_order_acquire;
    #define FENCE_0() 
    #define FENCE_1() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE_2()
#elif defined(THREAD_2_FENCE_ACQ)
    cuda::memory_order thread_0_store = cuda::memory_order_release;
    cuda::memory_order thread_1_load = cuda::memory_order_acquire;
    cuda::memory_order thread_1_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_2_load = cuda::memory_order_relaxed;
    #define FENCE_0() 
    #define FENCE_1()
    #define FENCE_2() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
#elif defined(THREAD_2_FENCE_REL)
    cuda::memory_order thread_0_store = cuda::memory_order_release;
    cuda::memory_order thread_1_load = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_store = cuda::memory_order_release;
    cuda::memory_order thread_2_load = cuda::memory_order_relaxed;
    #define FENCE_0() 
    #define FENCE_1()
    #define FENCE_2() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
#elif defined(THREAD_0_1_FENCE)
    cuda::memory_order thread_0_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_load = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_2_load = cuda::memory_order_acquire;
    #define FENCE_0() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE_1() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE_2() 
#elif defined(THREAD_0_2_FENCE_ACQ)
    cuda::memory_order thread_0_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_load = cuda::memory_order_acquire;
    cuda::memory_order thread_1_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_2_load = cuda::memory_order_relaxed;
    #define FENCE_0() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE_1() 
    #define FENCE_2() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
#elif defined(THREAD_0_2_FENCE_REL)
    cuda::memory_order thread_0_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_load = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_store = cuda::memory_order_release;
    cuda::memory_order thread_2_load = cuda::memory_order_relaxed;
    #define FENCE_0() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE_1() 
    #define FENCE_2() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
#elif defined(THREAD_1_2_FENCE)
    cuda::memory_order thread_0_store = cuda::memory_order_release;
    cuda::memory_order thread_1_load = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_2_load = cuda::memory_order_relaxed;
    #define FENCE_0() 
    #define FENCE_1() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
    #define FENCE_2() cuda::atomic_thread_fence(cuda::memory_order_acq_rel, FENCE_SCOPE);
#else
    cuda::memory_order thread_0_store = cuda::memory_order_relaxed; // default to all relaxed
    cuda::memory_order thread_1_load = cuda::memory_order_relaxed;
    cuda::memory_order thread_1_store = cuda::memory_order_relaxed;
    cuda::memory_order thread_2_load = cuda::memory_order_relaxed;
    #define FENCE_0()
    #define FENCE_1()
    #define FENCE_2()
#endif

    // defined for different distributions of threads across threadblocks
    DEFINE_IDS();

    THREE_THREAD_THREE_MEM_LOCATIONS();

    if (kernel_params->pre_stress) {
      do_stress(scratchpad, scratch_locations, kernel_params->pre_stress_iterations, kernel_params->pre_stress_pattern);
    }
    if (kernel_params->barrier) {
      spin(barrier, blockDim.x * kernel_params->testing_workgroups);
    }

    if (id_0 != id_1 && id_1 != id_2 && id_0 != id_2) {

      // Thread 0
      test_locations[x_0].store(1, cuda::memory_order_relaxed);
      FENCE_0()
      test_locations[y_0].store(1, thread_0_store);

      // Thread 1
      uint r0 = test_locations[y_1].load(thread_1_load);
      FENCE_1()
      test_locations[z_1].store(1, thread_1_store);

      // Thread 2
      uint r1 = test_locations[z_2].load(thread_2_load);
      FENCE_2()
      uint r2 = test_locations[x_2].load(cuda::memory_order_relaxed);

      cuda::atomic_thread_fence(cuda::memory_order_seq_cst);
      read_results[wg_offset + id_1].r0 = r0;
      read_results[wg_offset + id_2].r1 = r1;
      read_results[wg_offset + id_2].r2 = r2;
    }
  }
  else if (kernel_params->mem_stress) {
    do_stress(scratchpad, scratch_locations, kernel_params->mem_stress_iterations, kernel_params->pre_stress_iterations);
  }
}

__global__ void check_results(
  d_atomic_uint* test_locations,
  ReadResults* read_results,
  TestResults* test_results,
  KernelParams* kernel_params,
  bool* weak) {
  uint id_0 = blockIdx.x * blockDim.x + threadIdx.x;
  uint r0 = read_results[id_0].r0;
  uint r1 = read_results[id_0].r1;
  uint r2 = read_results[id_0].r2;
  uint x = test_locations[id_0 * kernel_params->mem_stride * 3];

  if (x == 0) {
    test_results->na.fetch_add(1); // thread skipped
  }
  else if (r0 == 1 && r1 == 1 && r2 == 1) {
    test_results->res0.fetch_add(1);
  }
  else if (r0 == 0 && r1 == 0 && r2 == 0) {
    test_results->res1.fetch_add(1);
  }
  else if (r0 == 0 && r1 == 0 && r2 == 1) {
    test_results->res2.fetch_add(1);
  }
  else if (r0 == 0 && r1 == 1 && r2 == 0) {
    test_results->res3.fetch_add(1);
  }
  else if (r0 == 0 && r1 == 1 && r2 == 1) {
    test_results->res4.fetch_add(1);
  }
  else if (r0 == 1 && r1 == 0 && r2 == 0) {
    test_results->res5.fetch_add(1);
  }
  else if (r0 == 1 && r1 == 0 && r2 == 1) {
    test_results->res6.fetch_add(1);
  }
  else if (r0 == 1 && r1 == 1 && r2 == 0) {
    test_results->weak.fetch_add(1);
    weak[id_0] = true;
  }
  else {
    test_results->other.fetch_add(1);
  }
}

int host_check_results(TestResults* results, bool print) {
  if (print) {
    std::cout << "r0=0, r1=1, r2=1 (seq): " << results->res0 << "\n";
    std::cout << "r0=0, r1=0, r2=0 (seq): " << results->res1 << "\n";
    std::cout << "r0=0, r1=0, r2=1 (seq): " << results->res2 << "\n";
    std::cout << "r0=0, r1=1, r2=0: " << results->res3 << "\n";
    std::cout << "r0=0, r1=1, r2=1: " << results->res4 << "\n";
    std::cout << "r0=1, r1=0, r2=0 (seq): " << results->res5 << "\n";
    std::cout << "r0=1, r1=0, r2=1 (interleaved): " << results->res6 << "\n";
    std::cout << "r0=1, r1=1, r2=0 (weak): " << results->weak << "\n";
    std::cout << "thread skipped: " << results->na << "\n";
    std::cout << "other: " << results->other << "\n";
  }
  return results->weak;
}

