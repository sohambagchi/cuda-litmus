#include "hip/hip_runtime.h"
#include <iostream>
#include "litmus.cuh"
#include "functions.cu"

__global__ void litmus_test(
  d_atomic_uint* test_locations,
  ReadResults* read_results,
  uint* shuffled_workgroups,
  cuda::atomic<uint, cuda::thread_scope_device>* barrier,
  uint* scratchpad,
  uint* scratch_locations,
  KernelParams* kernel_params,
  TestInstance* test_instances) {

  uint shuffled_workgroup = shuffled_workgroups[blockIdx.x];
  if (shuffled_workgroup < kernel_params->testing_workgroups) {

#ifdef ACQ_REL 
    cuda::memory_order store_order = cuda::memory_order_release;
    cuda::memory_order load_order = cuda::memory_order_acquire;
#elif defined(RELAXED)
    cuda::memory_order store_order = cuda::memory_order_relaxed;
    cuda::memory_order load_order = cuda::memory_order_relaxed;
#else
    cuda::memory_order store_order = cuda::memory_order_relaxed;
    cuda::memory_order load_order = cuda::memory_order_relaxed;
#endif

    // defined for different distributions of threads across threadblocks
    DEFINE_IDS();

    // defined for all three thread three memory locations tests
    THREE_THREAD_THREE_MEM_LOCATIONS();

    PRE_STRESS();

    if (id_0 != id_1 && id_1 != id_2 && id_0 != id_2) {

      // Thread 0
      test_locations[x_0].store(1, store_order);
      test_locations[y_0].store(1, store_order);

      // Thread 1
      test_locations[y_1].store(2, store_order);
      test_locations[z_1].store(1, store_order);

      // Thread 2
      uint r0 = test_locations[z_2].load(load_order);
      uint r1 = test_locations[x_2].load(load_order);

      cuda::atomic_thread_fence(cuda::memory_order_seq_cst);
      read_results[wg_offset + id_2].r0 = r0;
      read_results[wg_offset + id_2].r1 = r0;
    }
  }

  MEM_STRESS();
}

__global__ void check_results(
  d_atomic_uint* test_locations,
  ReadResults* read_results,
  TestResults* test_results,
  KernelParams* kernel_params,
  bool* weak) {
  RESULT_IDS();
  uint id_0 = blockIdx.x * blockDim.x + threadIdx.x;
  uint r0 = read_results[id_0].r0;
  uint r1 = read_results[id_0].r1;
  uint x = test_locations[id_0 * kernel_params->mem_stride * 3];
  uint y_loc = (wg_offset + permute_id(id_0, kernel_params->permute_location, total_ids)) * kernel_params->mem_stride * 3 + kernel_params->mem_offset;
  uint y = test_locations[y_loc];

  if (x == 0) {
    test_results->na.fetch_add(1); // thread skipped
  }
  else if (y == 2 && r0 == 1 && r1 == 0) {
    test_results->weak.fetch_add(1);
    weak[id_0] = true;
  }
  else {
    test_results->other.fetch_add(1);
  }
}

int host_check_results(TestResults* results, bool print) {
  if (print) {
    std::cout << "r0=1, r1=0, y=2 (weak): " << results->weak << "\n";
    std::cout << "thread skipped: " << results->na << "\n";
    std::cout << "other: " << results->other << "\n";
  }
  return results->weak;
}

